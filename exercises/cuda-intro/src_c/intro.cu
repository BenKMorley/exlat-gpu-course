/*
 * This is a simple CUDA code that negates an array of integers.
 * It introduces the concepts of device memory management, and
 * kernel invocation.
 *
 * Training material developed by James Perry and Alan Gray
 * Copyright EPCC, The University of Edinburgh, 2010 
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

/* Utility function to check for and report CUDA errors */
void checkCUDAError(const char*);

/* The number of integer elements in the array */
#define ARRAY_SIZE 256

/*
 * The number of CUDA blocks and threads per block to use.
 * These should always multiply to give the array size.
 * For the single block kernel, NUM_BLOCKS should be 1 and
 * THREADS_PER_BLOCK should be the array size
 */

#define NUM_BLOCKS  1
#define THREADS_PER_BLOCK 256

/* The actual array negation kernel (basic single block version) */
__global__ void negate(int * d_a) {
  /* Part 2B: negate an element of d_a */
  int i = threadIdx.x;
  d_a[i] = -1.0 * d_a[i];
}

/* Multi-block version of kernel for part 2C */
#define NUM_BLOCKS  4
#define THREADS_PER_BLOCK 64

__global__ void negate_multiblock(int *d_a) {
  /* Part 2C: negate an element of d_a, using multiple blocks this time */
  int i = blockIdx.x;
  int j = threadIdx.x;
  int index = i * blockDim.x + j;

  d_a[index] = -d_a[index];
}

/* Main routine */

int main(int argc, char *argv[]) {
  int *h_a, *h_out;
  int *d_a;

  int i;
  size_t sz = ARRAY_SIZE * sizeof(int);

  /* Print device details */
  int deviceNum;
  hipGetDevice(&deviceNum);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, deviceNum);
  printf("Device name: %s\n", prop.name);

  /* h_a holds the input array, h_out holds the result */
  h_a = (int *) malloc(sz);
  h_out = (int *) malloc(sz);

  /* Part 1A: allocate device memory */
  hipMalloc(&d_a, sz);

  /* initialise host arrays */
  for (i = 0; i < ARRAY_SIZE; i++) {
    h_a[i] = i;
    h_out[i] = 0;
  }

  printf("Hello World 1");

  /* Part 1B: copy host array h_a to device array d_a */
  hipMemcpy(h_a, d_a, sz * sizeof(int), hipMemcpyDeviceToHost);

  /* Part 2A: configure and launch kernel (un-comment and complete) */
  dim3 blocksPerGrid(1, 1, 1);
  dim3 threadsPerBlock(256, 1, 1);
  negate <<<blocksPerGrid, threadsPerBlock>>>(d_a);

  /* wait for all threads to complete and check for errors */
  hipDeviceSynchronize();
  checkCUDAError("kernel invocation");

  /* Part 1C: copy device array d_a to host array h_out */
  hipMemcpy(h_out, d_a, sz * sizeof(int), hipMemcpyHostToDevice);
  checkCUDAError("memcpy");

  /* print out the result */
  printf("Results: ");
  for (i = 0; i < ARRAY_SIZE; i++) {
    printf("%d, ", h_out[i]);
  }

  printf("\n\n");

  /* Part 1D: free d_a */
  hipFree(d_a);

  /* free host buffers */
  free(h_a);
  free(h_out);

  return 0;
}

/* Utility function to check for and report CUDA errors */

void checkCUDAError(const char * msg) {

  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}
