#include "hip/hip_runtime.h"
/*
 * Skeleton for Very Basic Linear Solver Development
 *
 * Nick Johnson, EPCC && ExaLAT.
 */

#include <stdio.h>
#include <stdlib.h>

/* Utility function to check for and report CUDA errors */
void checkCUDAError(const char*);

/* Utility Function to enumerate devices from SLURM/PBS etc */
void deviceEnumerator(int *cuda_devices, int *cuda_device_count);

/* The number of integer elements in the array */
#define ARRAY_SIZE 32

/*
 * The number of CUDA blocks and threads per block to use.
 * These should always multiply to give the array size.
 * For the single block kernel, NUM_BLOCKS should be 1 and
 * THREADS_PER_BLOCK should be the array size
 */
#define NUM_BLOCKS 32
#define THREADS_PER_BLOCK 32

/* Define max number of devices we expect per node. It's currently 8 on Cirrus, so we keep to that for now. */
#define MAX_DEVICES 8


/*
 * Matrix Vector product
 * input:  matrix, pointer to a previously allocated 1D matrix
 * input:  vector, pointer to a previously allocated vector
 * output: resvector, pointer to a previously allocated vector which will contain the result
 */
__global__ void matrix_vector(float *matrix, float *vector, float *resvector) {

  /*
   * Implement Matrix Vector here
   */
  
}

/*
 * Vector Vector product (dot product)
 * input:  vectorA, pointer to a previously allocated vector
 * input:  vectorB, pointer to a previously allocated vector
 * output: resscalar, pointer to a previously allocated scalar which will contain the result
 */
__global__ void vector_vector(float *vectorA, float *vectorB, float *resscalar) {

  /*
   * Implement your vector dot product here
   */  

}

/*
 * Vector plus Vector
 * input:  vectorA, pointer to a previously allocated vector
 * input:  vectorB, pointer to a previously allocated vector
 * output: resvector, pointer to a previously allocated vector which will contain the result
 */
__global__ void vector_add(float *vectorA, float *vectorB, float *resvector) {

  /*
   * Implement your vector add here
   */   
  
}

/*
 * Vector plus Factor * Vector
 * input:  vectorA, pointer to a previously allocated vector
 * input:  vectorB, pointer to a previously allocated vector
 * input:  factor, a scalar which elementwise multiplies the second vector
 * output: resscalar, pointer to a previously allocated vector which will contain the result
 */
__global__ void vector_add_factor(float *vectorA, float *vectorB, float factor, float *resvector) {

  /*
   * Implement your vector multi with factor here
   */
  
}




/*
 * Vector minus Vector
 * input:  vectorA, pointer to a previously allocated vector
 * input:  vectorB, pointer to a previously allocated vector
 * output: resscalar, pointer to a previously allocated vector which will contain the result
 */
__global__ void vector_minus(float *vectorA, float *vectorB, float *resvector) {

  /*
   * Implement your vector subtraction here
   */
  
}

/*
 * Vector minus Factor * Vector
 * input:  vectorA, pointer to a previously allocated vector
 * input:  vectorB, pointer to a previously allocated vector
 * input:  factor, a scalar which multiplies the second vector elementwise
 * output: resscalar, pointer to a previously allocated vector which will contain the result
 */
__global__ void vector_minus_factor(float *vectorA, float *vectorB, float factor, float *resvector) {

  /*
   * Implement your vector subtraction with factor here
   */
  
}

/*
 * Function which seeds a square matrix of ARRAY_SIZE x ARRAY_SIZE with positive values on the leading diag.
 */
int seedvectors(float *matrix){

  int i = 0;
  int j = 0;

  for (j = 0; j < ARRAY_SIZE; j++){
    for (i = 0; i< ARRAY_SIZE; i++){
      if (i == j){
        matrix[j*ARRAY_SIZE +i] = (float)rand()/RAND_MAX;
      }
      else{
        matrix[j*ARRAY_SIZE +i] = 0;
      }

    }
  }

  return 0;
}



/* Main function */

int main(int argc, char *argv[]) {

  /*
   * This is pre-amble code to deal with multiple GPUs, please do not edit.
   */

  /*
   * cuda_devices holds the handles of the cuda devices we need to pass around in a parallel case
   */
  int *cuda_devices = (int*) calloc(MAX_DEVICES, sizeof(int));
  int cuda_device_count = 0;

  
  /*
   * Check that there are some GPUs, but not too many
   */
  deviceEnumerator(cuda_devices, &cuda_device_count);
  if (cuda_device_count == -1 || cuda_device_count > MAX_DEVICES){
    printf("Error enumerating CUDA devices - found %d.\n Exiting.\n", cuda_device_count);
    return 1;
  }


  /*
   * We print out the properties of the CUDA devices (GPUs in this case, but could be CPUs etc)
   * This is useful to know, but also servies as a quick check we can access the devices
   */
  int i = 0;
  int cuda_k = 0;
  int deviceNum = 0;
  hipDeviceProp_t prop;

  printf("Number of CUDA Devices = %d\n", cuda_device_count);
  for (i = 0; i < cuda_device_count; i++){
    deviceNum = cuda_devices[i];
    hipGetDeviceProperties(&prop, deviceNum);
    printf("\tDevice %d : Device name: %s\n", deviceNum, prop.name);
    
  }
  printf("\n");



  /*
   * End pre-amble
   */


  /*
   * Begin main code
   */


  /*
   * Some useful helper sizes and variables
   */
  int j = 0;
  size_t matrix_sz = ARRAY_SIZE * ARRAY_SIZE * sizeof(float);
  size_t vector_sz = ARRAY_SIZE * sizeof(float);
  size_t scalar_sz = 1 * sizeof(float);
  float scalar = 0;

  

  /*
   * Create pointers to hold data on the host
   */
  float *matrixA = NULL;
  float *vectorR = NULL;
  float *vectorB = NULL;
  float *vectorX = NULL;
  float *vectorP = NULL;

  float *vectorRnew = NULL;
  float *vectorXnew = NULL;
  float *vectorPnew = NULL;

  /*
   * Allocate memory on host & test it was successful
   * This is an often missed step and can catch you out
   * We use heap allocations rather than stack for two reasons
   * 1. It makes everything a pointer which marries nicely with hipMalloc
   * 2. It would be easy to fill the stack space and we cannot use ulimit on all systems to increase it
   */
  matrixA = (float *) calloc(matrix_sz, 1);
  vectorR = (float *) calloc(vector_sz, 1);
  vectorB = (float *) calloc(vector_sz, 1);
  vectorP = (float *) calloc(vector_sz, 1);
  vectorX = (float *) calloc(vector_sz, 1);
  vectorRnew = (float *) calloc(vector_sz, 1);
  vectorXnew = (float *) calloc(vector_sz, 1);
  vectorPnew = (float *) calloc(vector_sz, 1);
  
  if (matrixA == NULL ||\
      vectorR == NULL ||\
      vectorB == NULL ||\
      vectorP == NULL ||\
      vectorX == NULL ||\
      vectorRnew == NULL ||\
      vectorXnew == NULL ||\
      vectorPnew == NULL){
    printf("Error allocating host memory.\n");
    return 1;
  }

  /*
   * Initialise host arrays
   * Calloc should push these to be 0, but using this method we can pick anything.
   * Having a non-zero initialiser for the output array can help spot problems if we never expect a 0 in the output
   */
  seedvectors(matrixA);
  for (j = 0; j < ARRAY_SIZE; j++){
    vectorP[j] = 0;
    vectorB[j] = 1.0;
    vectorX[j] = 0;
    vectorRnew[j] = 0;
    vectorXnew[j] = 0;
    vectorPnew[j] = 0; 
  }



  
  /*
   * Create pointers to hold data on the device
   */  
  float *device_matrixA = NULL;
  float *device_vectorR = NULL;
  float *device_vectorB = NULL;
  float *device_vectorX = NULL;
  float *device_vectorP = NULL;
  
  float *device_vectorRnew = NULL;
  float *device_vectorXnew = NULL;
  float *device_vectorPnew = NULL;
  float *device_vectorAP = NULL;
    
  float *device_scalar = NULL;



  /*
   * Set up some useful values
   * threadsPerBlock is as in previous exercises and simply a cast from the macro define
   * nBlocks is split across the number of devices we have
   */
  dim3 threadsPerBlock(THREADS_PER_BLOCK);
  dim3 nBlocks(NUM_BLOCKS/cuda_device_count);


  printf("numBlocks: %d\n", (NUM_BLOCKS/cuda_device_count));
  printf("threadsPerBlock: %d\n", THREADS_PER_BLOCK);

  /*
   * The compiler ignores pragmas statements which it cannot parse, so this can live outside the guard
   * The value of cuda_k will be 0 for a serial case so we get a single iteration of this loop
   * and ergo a single thread of execution.
   */

  for (cuda_k = 0; cuda_k < cuda_device_count; cuda_k++) {
   
    deviceNum = cuda_devices[cuda_k];
    hipSetDevice(deviceNum);
    hipGetDeviceProperties(&prop, deviceNum);
    
    printf("%d:  Device name: %s\n", deviceNum, prop.name);
  
    /*
     * Allocate device memory
     * This is done inside the loop to give us some flexibility in a multi-GPU case
     */
    hipMalloc(&device_matrixA, matrix_sz/cuda_device_count);
    checkCUDAError("Device matrixA allocation");
    hipMalloc(&device_vectorR, vector_sz);
    checkCUDAError("Device vectorR allocation");
    hipMalloc(&device_vectorB, vector_sz);
    checkCUDAError("Device vectorB allocation");
    hipMalloc(&device_vectorP, vector_sz);
    checkCUDAError("Device vectorP allocation");
    hipMalloc(&device_vectorX, vector_sz);
    checkCUDAError("Device vectorX allocation");

    hipMalloc(&device_vectorRnew, vector_sz);
    checkCUDAError("Device vectorRnew allocation");
    hipMalloc(&device_vectorPnew, vector_sz);
    checkCUDAError("Device vectorPnew allocation");
    hipMalloc(&device_vectorXnew, vector_sz);
    checkCUDAError("Device vectorXnew allocation");

    hipMalloc(&device_vectorAP, vector_sz);
    checkCUDAError("Device vectorAP allocation");

    hipMalloc(&device_scalar, scalar_sz);
    checkCUDAError("Device vectorXnew allocation");



    /*
     * This is the start of the initialisation step
     * We must derive an initial R_0, compute Rs and set P = R_0
     */

    
    /*
     * Copy arrays and matrices to device(s)
     * The offset arrangement helps with >1 GPU
     */
    hipMemcpy(device_matrixA, matrixA, matrix_sz, hipMemcpyHostToDevice);
    checkCUDAError("Memcpy: H2D matrix");
    hipMemcpy(device_vectorX, vectorX, vector_sz, hipMemcpyHostToDevice);
    checkCUDAError("Memcpy: H2D vectorX");

    /*
     * Compute the first step Ax and save somewhere (Xnew??)
     */
    matrix_vector<<<>>>();
   
    /*
     * Compute the second step b - (Ax) and put in r
     */
    hipMemcpy(device_vectorB, vectorB, vector_sz, hipMemcpyHostToDevice);
    vector_minus<<<>>>();

    /*
     * Copy this back to the host
     */
    hipMemcpy(vectorR, device_vectorR, vector_sz, hipMemcpyDeviceToHost);
   
  
    
    // Since p = r, copy this initial r to device p host side only!
    memcpy(vectorP, vectorR, vector_sz);

    

    
    scalar = 0;
    hipMemcpy(device_scalar, &scalar, scalar_sz, hipMemcpyHostToDevice);
    checkCUDAError("Memcpy: H2D scalar");
    hipDeviceSynchronize();

    /*
     * Compute R . R into device_scalar
     */
    vector_vector<<<>>>();
    
    
    float initial_rs = scalar;
    printf("Initial Rs = %f\n", initial_rs);
    float rsold = initial_rs;
    float beta = 0;
    float alpha = 0;
    float rsnew = 0;

    
    /*
     * This is the end of the initialisation step
     * We have derived an initial R_0, computed Rs and set P = R_0
     */


    /*
     * This is the start of the main loop
     * We now need to compute alpha, then R_k+1, beta, P_k+1 etc.
     * Once we have computed the value of (R_k+1)s, ie the updated residual, we can stop.
     */   
    int mainloop = 0;
    for (mainloop = 0; mainloop < ARRAY_SIZE; mainloop++){
    
      /*
       * Compute A times P and store, temporarily in Pnew
       */
      matrix_vector<<<>>>();

            /*
       * Compute Ap dot P
       */
      vector_vector<<<>>>();



      /*
       * Compute Alpha
       */
      alpha = 0;
      alpha = rsold / scalar;

      /*
       * Compute x_k+1 = x_k + alpha.*P_k
       * Store in Xnew
       */
      vector_add_factor<<<>>>():

      /*
       * Compute R_k+1 = R_k - alpha.*(AP_k)
       * Store in Rnew
       */
      vector_minus_factor<<<>>>():
  

      // Calculate Beta
      // Rnew dot Rnew / R dot R
      scalar = 0;
      vector_vector<<<>>>();
      rsnew = 0;

    
    
      beta = rsnew / rsold;

      // Make Pnew = Rnew + Beta P   
      vector_add_factor<<<>>>();
   
 
    
      /*
       * Set up for next iteration
       */
      rsold = rsnew;
      memcpy(vectorP, vectorPnew, vector_sz);
      memcpy(vectorR, vectorRnew, vector_sz);
      memcpy(vectorX, vectorXnew, vector_sz);
      
    }
    
    /*
     * Herein lies the end of the first iteration.
     */
    
    

    /*
     * Free the device memory
     */
    hipFree(device_matrixA);
    hipFree(device_vectorR);
    hipFree(device_vectorB);
    hipFree(device_vectorP);
    hipFree(device_vectorX);
    hipFree(device_vectorRnew);
    hipFree(device_vectorPnew);
    hipFree(device_vectorXnew);
    hipFree(device_vectorAP);
    hipFree(device_scalar);
    
  } // end of cuda_k loop


  /*
   * Print the output vector and then free the host memory
   */
  // for (i = 0; i < ARRAY_SIZE; i++) {
  //   printf("%d ", host_result_vector[i]);
  // }
  // printf("\n\n");

  // int sum = 0;
  // for (i = 0; i < ARRAY_SIZE; i++){
  //   sum += vectorR[i] * vectorR[i];
  // }
  

  // printf("Scalar = %d\n", scalar);
  // printf("Sum = %d\n", sum);
  

  free(matrixA);
  free(vectorR);
  free(vectorB);
  free(vectorP);
  free(vectorX);
  free(vectorRnew);
  free(vectorPnew);
  free(vectorXnew);


  // Fin.
  return 0;
}


/* Utility function to check for and report CUDA errors */
void checkCUDAError(const char * msg) {

  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*
 * Print device details
 *
 * Based on code available at:
 * https://wiki.sei.cmu.edu/confluence/display/c/STR06-C.+Do+not+assume+that+strtok%28%29+leaves+the+parse+string+unchanged
 */
void deviceEnumerator(int *cuda_devices, int *cuda_device_count){

  char * tokenized = NULL;

  /*
   * CUDA_VISIBLE_DEVICES may not be present on all systems
   * Or, another env. variable might be used
   * This works for Cirrus at EPCC
   */
  const char* s = getenv("CUDA_VISIBLE_DEVICES");

  // If s is NULL, CUDA_VISIBLE_DEVICES was empty, ie unset and, on productions systems, means no GPUs available
  if (s == NULL){
    *cuda_device_count = -1; // Flag as error
    return;
  }

  // If we cannot allocate enough memory here, something has gone wrong
  char * copy = (char *) malloc(strlen(s) + 1);
  if (copy == NULL) {
    *cuda_device_count = -1; // Flag as error
    return;
  }


  /*
   * Iterate over a copy of s (called copy) and look for device handles
   * Store those in the cuda_devices array, and increment the device count local_cdc
   * Return local_cdc as cuda_device_count
   */
  int local_cdc = 0; 
  strcpy(copy, s);
  
  tokenized = strtok(copy, ",");
  *cuda_devices++ = atoi(tokenized);
  local_cdc++;

  while (tokenized = strtok(0, ",")) {
    *cuda_devices++ = atoi(tokenized);
    local_cdc++;
  }

  *cuda_device_count = local_cdc;

}